#include "hip/hip_runtime.h"
#include "pgmProcess.h"

/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distance( int p1[], int p2[] )
{
	float sqr = (float) (p1[0] - p2[0]) * (p1[0] - p2[0]) + (p1[1] - p2[1]) * (p1[1] - p2[1]);
	return sqrtf(sqr);
}

__global__ void drawCircle(int* pixels, int dimx, int dimy, int centerCol, int centerRow, int radius)
{
	int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    	int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    	int idx = iy*dimx + ix;
    	
    	int center[2] = {centerCol, centerRow};
    	int pixel[2] = {ix,iy};
    	
    	float distance = distance(center, pixel);
    	
    	if(distance <= radius && ix < dimx)
    	{
    		pixels[idx] = 0;
    	}
    	
}
__global__ void drawEdge(int* pixels, int dimx, int dimy, int edgeWidth)
{
        int x = threadIdx.x+(blockIdx.x*blockDim.x);
        int y = threadIdx.y+(blockIdx.y*blockDim.y);

	if(x<xmax && y < ymax)
	{
		if(x<size||y<size||(ymax-y)<=size||(xmax-x)<=size)
			arr[y*xmax + x] = 0;
	}
}

__global__ void pgmDrawLineKernel(int *pixels, int *indices, int dimx, int numIndices) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int thread = ix * dimx + iy;
    int index = -1;
    if (thread < numIndices) index = indices[thread];
    if (index != -1) pixels[index] = 0;
}
