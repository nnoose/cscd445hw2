#include "hip/hip_runtime.h"


/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distance( int p1[], int p2[] )
{
	float sqr = (float)(p1[0]-p2[0])(p1[0]-p2[0]) + (p1[1] - p2[1])(p1[1]-p2[1]);
	return sqrtf(sqr);
}
