#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "pgmProcess.h"
#include "pgmUtility.h"
// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.

int * pgmRead( char **header, int *numRows, int *numCols, FILE *in )
{
    int i, j;
    // read in header of the image first
    for( i = 0; i < rowsInHeader; i ++)
    {
        if ( header[i] == NULL )
        {
            return NULL;
        }
        if( fgets( header[i], maxSizeHeadRow, in ) == NULL )
        {
            return NULL;
        }
    }
    // extract rows of pixels and columns of pixels
    sscanf( header[rowsInHeader - 2], "%d %d", numCols, numRows );  // in pgm the first number is # of cols

    // Now we can intialize the pixel of 2D array, allocating memory
    int *pixels = ( int * ) malloc( ( *numRows ) * ( *numCols ) * sizeof( int ) );

    // read in all pixels into the pixels array.
    for( i = 0; i < *numRows; i ++ )
        for( j = 0; j < *numCols; j ++ )
            if ( fscanf(in, "%d ", &pixels[i*(*numCols)+j]) < 0 )
                return NULL;

    return pixels;
}

int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out )
{
    int i, j;

    // write the header
    for ( i = 0; i < rowsInHeader; i ++ )
    {
        fprintf(out, "%s", *( header + i ) );
    }

    // write the pixels
    for( i = 0; i < numRows; i ++ )
    {
        for ( j = 0; j < numCols; j ++ )
        {
            if ( j < numCols - 1 )
                fprintf(out, "%d ", pixels[i*numCols + j]);
            else
                fprintf(out, "%d\n", pixels[i*numCols+j]);
        }
    }
    return 0;
}



int pgmDrawEdge( int *pixels, int numRows, int numCols, int edgeWidth, char **header )
{

	//init variables. Threads x/y can be defined in header
	int threadsX = 32;
	int threadsY = 32;
	int blocksX = ceil(numCols/(float)threadsX);
	int blocksY = ceil(numRows/(float)threadsY);
	dim3 grid(blocksX,blocksY,1);
	dim3 block(threadsX,threadsY,1);

	//alloc device array
	int * arr;
	hipMalloc(&arr,numRows*numCols*sizeof(int));

	hipMemcpy(arr,pixels, numRows*numCols*sizeof(int), hipMemcpyHostToDevice);

	//call kernel
	makeEdge<<<grid,block>>>(arr,numCols,numRows,edgeWidth);

	hipMemcpy(pixels,arr, numRows*numCols*sizeof(int), hipMemcpyDeviceToHost);


	return 0;
}


