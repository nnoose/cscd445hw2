#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include "pgmProcess.h"
#include "pgmUtility.h"
// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.

int * pgmRead( char **header, int *numRows, int *numCols, FILE *in )
{
    int i, j;
    // read in header of the image first
    for( i = 0; i < rowsInHeader; i ++)
    {
        if ( header[i] == NULL )
        {
            return NULL;
        }
        if( fgets( header[i], maxSizeHeadRow, in ) == NULL )
        {
            return NULL;
        }
    }
    // extract rows of pixels and columns of pixels
    sscanf( header[rowsInHeader - 2], "%d %d", numCols, numRows );  // in pgm the first number is # of cols

    // Now we can intialize the pixel of 2D array, allocating memory
    int *pixels = ( int * ) malloc( ( *numRows ) * ( *numCols ) * sizeof( int ) );

    // read in all pixels into the pixels array.
    for( i = 0; i < *numRows; i ++ )
        for( j = 0; j < *numCols; j ++ )
            if ( fscanf(in, "%d ", &pixels[i*(*numCols)+j]) < 0 )
                return NULL;

    return pixels;
}

int pgmWrite( const char **header, const int *pixels, int numRows, int numCols, FILE *out )
{
    int i, j;

    // write the header
    for ( i = 0; i < rowsInHeader; i ++ )
    {
        fprintf(out, "%s", *( header + i ) );
    }

    // write the pixels
    for( i = 0; i < numRows; i ++ )
    {
        for ( j = 0; j < numCols; j ++ )
        {
            if ( j < numCols - 1 )
                fprintf(out, "%d ", pixels[i*numCols + j]);
            else
                fprintf(out, "%d\n", pixels[i*numCols+j]);
        }
    }
    return 0;
}



int pgmDrawEdge( int *pixels, int numRows, int numCols, int edgeWidth, char **header )
{

	//init variables. Threads x/y can be defined in header
	clock_t start, end;
	int threadsX = 32;
	int threadsY = 32;
	int blocksX = ceil(numCols/(float)threadsX);
	int blocksY = ceil(numRows/(float)threadsY);
	dim3 grid(blocksX,blocksY,1);
	dim3 block(threadsX,threadsY,1);

	//alloc device array
	int * arr;
	hipMalloc(&arr,numRows*numCols*sizeof(int));

	start = clock();
	hipMemcpy(arr,pixels, numRows*numCols*sizeof(int), hipMemcpyHostToDevice);

	//call kernel
	makeEdge<<<grid,block>>>(arr,numCols,numRows,edgeWidth);

	hipMemcpy(pixels,arr, numRows*numCols*sizeof(int), hipMemcpyDeviceToHost);
	end = clock();
	
	double totalTime = (end-start)/CLOCKS_PER_SEC;
    	printf("Total GPU time taken for Edge: %f\n", totalTime);


	return 0;
}

int pgmDrawCircle( int *pixels, int numRows, int numCols, int centerRow, int centerCol, int radius, char **header )
{
	clock_t start, end;
	int threadsX = 32;
	int threadsY = 32;
	int blocksX = ceil(numCols/(float)threadsX);
	int blocksY = ceil(numRows/(float)threadsY);
	dim3 grid(blocksX,blocksY,1);
	dim3 block(threadsX,threadsY,1);
	
	int* d_in =0;
	
    	int byteSize = sizeof(pixels)/sizeof(int);
    	
    	start = clock();
    	hipMemcpy(d_in, pixels, byteSize, hipMemcpyHostToDevice);
    	drawCircle<<<grid, block>>>(d_in, numCols, numRows, centerCol, centerRow, radius);
    	hipMemcpy(pixels, d_in, byteSize, hipMemcpyDeviceToHost);
    	end = clock();
    	
    	double totalTime = (end-start)/CLOCKS_PER_SEC;
    	printf("Total GPU time taken for Circle: %f\n", totalTime);
    
	return 0;
}


